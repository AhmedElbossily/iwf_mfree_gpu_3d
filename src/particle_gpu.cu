//Copyright ETH Zurich, IWF

//This file is part of iwf_mfree_gpu_3d.

//iwf_mfree_gpu_3d is free software: you can redistribute it and/or modify
//it under the terms of the GNU General Public License as published by
//the Free Software Foundation, either version 3 of the License, or
//(at your option) any later version.

//iwf_mfree_gpu_3d is distributed in the hope that it will be useful,
//but WITHOUT ANY WARRANTY; without even the implied warranty of
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//GNU General Public License for more details.

//You should have received a copy of the GNU General Public License
//along with mfree_iwf.  If not, see <http://www.gnu.org/licenses/>.

#include "particle_gpu.h"

particle_gpu::particle_gpu(unsigned int N) {
	hipMalloc((void **) &pos, sizeof(float4_t)*N);
	hipMalloc((void **) &vel, sizeof(float4_t)*N);
	hipMalloc((void **) &vel_bc, sizeof(float4_t)*N);

	hipMalloc((void **) &h,   sizeof(float_t)*N);
	hipMalloc((void **) &rho, sizeof(float_t)*N);
	hipMalloc((void **) &p,   sizeof(float_t)*N);

	hipMalloc((void **) &S,   sizeof(mat3x3_t)*N);
	hipMalloc((void **) &R,   sizeof(mat3x3_t)*N);
	hipMalloc((void **) &fc,  sizeof(float3_t)*N);
	hipMalloc((void **) &ft,  sizeof(float3_t)*N);
	hipMalloc((void **) &n,   sizeof(float3_t)*N);

	hipMalloc((void**) &fixed, sizeof(float_t)*N);
	hipMalloc((void**) &blanked, sizeof(float_t)*N);
	hipMalloc((void**) &tool_particle, sizeof(float_t)*N);

	hipMalloc((void**) &eps_pl, sizeof(float_t)*N);
	hipMalloc((void**) &eps_pl_dot, sizeof(float_t)*N);
	hipMalloc((void**) &T, sizeof(float_t)*N);

	hipMalloc((void **) &pos_t, sizeof(float4_t)*N);
	hipMalloc((void **) &vel_t, sizeof(float4_t)*N);
	hipMalloc((void **) &rho_t, sizeof(float_t)*N);
	hipMalloc((void **) &S_t,   sizeof(mat3x3_t)*N);
	hipMalloc((void **) &T_t,   sizeof(float_t)*N);

	hipMalloc((void **) &v_der, sizeof(mat3x3_t)*N);
	hipMalloc((void **) &S_der, sizeof(mat3x3_t)*N);

	hipMalloc((void **) &idx, sizeof(int)*N);
	hipMalloc((void **) &hash, sizeof(int)*N);

	thrust::device_ptr<int> t_idx(this->idx);
	thrust::sequence(t_idx, t_idx+N);

	hipMalloc((void**) &num_nbh, sizeof(int)*N);

	hipMemset(pos, 0, sizeof(float4_t)*N);
	hipMemset(vel,0, sizeof(float4_t)*N);
	hipMemset(vel_bc,0, sizeof(float4_t)*N);

	hipMemset(h,0,   sizeof(float_t)*N);
	hipMemset(rho,0, sizeof(float_t)*N);
	hipMemset(p,0,   sizeof(float_t)*N);

	hipMemset(S,0,   sizeof(mat3x3_t)*N);
	hipMemset(R,0,   sizeof(mat3x3_t)*N);
	hipMemset(fc,0,  sizeof(float3_t)*N);
	hipMemset(ft,0,  sizeof(float3_t)*N);
	hipMemset(n,0,   sizeof(float3_t)*N);

	hipMemset(fixed,0,  sizeof(float_t)*N);
	hipMemset(blanked,0,  sizeof(float_t)*N);
	hipMemset(tool_particle, 0,  sizeof(float_t)*N);

	hipMemset(eps_pl, 0, sizeof(float_t)*N);
	hipMemset(eps_pl_dot, 0, sizeof(float_t)*N);
	hipMemset(T, 0, sizeof(float_t)*N);

	hipMemset(pos_t,0, sizeof(float3_t)*N);
	hipMemset(vel_t,0, sizeof(float3_t)*N);
	hipMemset(rho_t,0, sizeof(float_t)*N);
	hipMemset(S_t,0,   sizeof(mat3x3_t)*N);
	hipMemset(T_t,0,   sizeof(float_t)*N);

	hipMemset(v_der, 0, sizeof(mat3x3_t)*N);
	hipMemset(S_der, 0, sizeof(mat3x3_t)*N);

	hipMemset(num_nbh, 0, sizeof(int)*N);

	this->N = N;
	this->N_init = N;
}

particle_gpu::particle_gpu(float4_t *pos, float4_t *vel_init, float_t *rho, float_t *h, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float4_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float4_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float4_t *pos, float4_t *vel_init, float_t *rho, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, N) {
	hipMemcpy(this->fixed, fixed, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float4_t *pos, float4_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, fixed, N) {
	hipMemcpy(this->T, T_init, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float4_t *pos, float4_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, float_t *tool_p, unsigned int N)
: particle_gpu(pos, vel_init, rho, T_init, h, fixed, N) {
	hipMemcpy(this->tool_particle, tool_p, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float4_t *pos, float4_t *vel_init, float_t *rho, float_t *h, mat3x3_t *S, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float4_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float4_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->S,   S,        sizeof(mat3x3_t)*N, hipMemcpyHostToDevice);
}
